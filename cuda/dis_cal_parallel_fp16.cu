
#include <hip/hip_runtime.h>
#include <iostream>   
#include <iomanip>                                                                                                    
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "hip/hip_fp16.h"

// float16 半精度计算 100万2048维向量，占显存4G
// 注意：精度降低可能导致计算结果错误

using namespace std;

const int D = 2048;
const int N1 = 10000; // 数据文件条数
const int D1 = 100; // 数据重复倍数，方便模拟海量数据
const unsigned long N = N1*D1;


__global__ void cal_dis(half *train_data, half *test_data, half *dis, int pitch)
{
	//long tid = blockIdx.x;
	unsigned long tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid<N)
	{
		half temp = 0.0;
		half sum = 0.0;
		for(int i=0;i<D;i++)
		{
			temp = __hsub(*((half*)((char*)train_data + tid * pitch) + i), test_data[i]);
			sum = __hadd(sum, __hmul(temp, temp));
		}
		dis[tid] = sum;
	}
}

void print(half *data)
{
	cout<<"training data:"<<endl;
	for(unsigned long i=0;i<N;i++)
	{
		for(int j=0;j<D;j++)
		{
			cout<< fixed << setprecision(8)<<__half2float(*(data+i*D+j))<<" ";
		}
		cout<<endl;
	}
}
 
void print(half *data, unsigned long n)
{
	for(unsigned long i=0;i<n;i++)
	{
		cout<< fixed << setprecision(8)<<__half2float(data[i])<<" ";
	}
	cout<<endl;
}


int read_data(half *data_set)
{
	float f1;
	const char s[2] = ",";
	char *token, *line;
	FILE *fp;
	half test[D];

	// 一个数字假设占20字符，目前是保留16位小数，一共18个字符
	line = (char *)malloc(20*D*sizeof(char)); 

	fp = fopen("../vector.data" , "r");
	if(fp == NULL) {
		perror("打开文件时发生错误");
		return(-1);
	}

	// 读N+1行，最后1行做测试
	for(int i=0;i<N1+1;i++) {
		if( fgets (line, 20*D*sizeof(char), fp)!=NULL ) {
			token = strtok(line, s);

			int j = 0;
			while (token != NULL)
			{
				f1 = atof(token);
				//printf("%.8f ", f1);
				*(data_set+i*D+j)=__float2half(f1*10.0); // 增加10倍的精度

				token = strtok(NULL, s);
				j++;
			}
			//puts("");

		} else {
			break;
		}
	}

	fclose(fp);

	free(line);

	for(int i=0;i<D;i++) test[i]=*(data_set+N1*D+i); // 保存测试向量

	for(int d=1;d<D1;d++){ // 复制数据
		for(int i=0;i<N1;i++){
			for(int j=0;j<D;j++){
				*(data_set+(N1*d+i)*D+j)= *(data_set+i*D+j);
			}
		}
	}

	for(int i=0;i<D;i++) *(data_set+N*D+i)=test[i]; // 恢复测试向量

	return 0;
}

int main()
{
	

	half *h_train_data, *h_test_data;
	half distance[N];
 
	half *d_train_data , *d_test_data , *d_dis;
 
	float time1, time2;

	//printf("%d %d %d\n", sizeof(float), sizeof(half2), sizeof(half));

	// 显示GPU资源
	int dev = 0;
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, dev);
    std::cout << "使用GPU device " << dev << ": " << devProp.name << std::endl;
    std::cout << "SM的数量：" << devProp.multiProcessorCount << std::endl;
    std::cout << "每个线程块的共享内存大小：" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
    std::cout << "每个线程块的最大线程数：" << devProp.maxThreadsPerBlock << std::endl;
    std::cout << "每个EM的最大线程数：" << devProp.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "每个EM的最大线程束数：" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
    //-----------

	hipEvent_t start1, stop1, stop2;
	hipEventCreate(&start1);
	hipEventCreate(&stop1); 
	hipEventCreate(&stop2); 

	cout<<"num= "<<N<<"\tdim= "<<D<<endl;

	h_train_data = (half*)malloc((N+1)*D*sizeof(half));
	if (h_train_data==NULL){
		puts("alloc memory fail!");
		exit(-1);
	}

	size_t pitch_d;
	size_t pitch_h = D * sizeof(half) ; 

	//allocate memory on GPU 
	hipMallocPitch( &d_train_data, &pitch_d, D*sizeof(half), N); 
	hipMalloc((void**)&d_test_data, D*sizeof(half));
	hipMalloc((void**)&d_dis, N*sizeof(half)); // d_ids[N] 存最小值

	//initialize training data
	read_data(h_train_data);
	//print(h_train_data);
 
	//initialize testing data
	h_test_data = h_train_data+D*N;
	//cout<<"testing data:"<<endl;
	//print(h_test_data,D);
 

	//copy training and testing data from host to device
	hipMemcpy2D(d_train_data, pitch_d, h_train_data, pitch_h, D*sizeof(half), N, hipMemcpyHostToDevice);
	hipEventRecord(start1, 0); // 批量数据复制进GPU的耗时，不计入，现实中会提前载入
	hipMemcpy(d_test_data, h_test_data, D*sizeof(half), hipMemcpyHostToDevice);
 
	// 定义kernel的执行配置
	dim3 blockSize(256);
	dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
	printf("grid size: %d\tblock size: %d\n", gridSize.x, blockSize.x);
	// 执行kernel
	cal_dis<<<gridSize, blockSize>>>(d_train_data,d_test_data,d_dis,pitch_d);

	//calculate the distance
	//cal_dis<<<N,1>>>(d_train_data,d_test_data,d_dis,pitch_d);
 
	//copy distance data from device to host
	hipMemcpy(distance, d_dis, N*sizeof(half), hipMemcpyDeviceToHost);

	hipEventRecord(stop1, 0);

	// 找最小值
	float minimum = __half2float(distance[0]);
	unsigned long min_pos = 0;
	for(unsigned long i=1;i<N;i++) {
		float tmp_dis = __half2float(distance[i]);
		if (tmp_dis<minimum) {
			minimum=tmp_dis;
			min_pos=i;
		}
	}

	hipEventRecord(stop2, 0);
 
	//cout<<"distance:"<<endl;
	//print(distance, N);

	hipFree(d_train_data);
	hipFree(d_test_data);
	hipFree(d_dis);
	free(h_train_data);
	
	printf("min= %.8f\tpos= %ld\n", minimum, min_pos);

	hipEventElapsedTime(&time1, start1, stop1);
	hipEventElapsedTime(&time2, stop1, stop2);
	printf("[ time taken: %fms %fms ]\n",time1, time2);


	return 0;
}  
