
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <iostream>   
#include <iomanip>                                                                                                    
#include <stdio.h>
#include <stdlib.h>
#include <time.h>


using namespace std;

const int D = 2048;
const int N1 = 10000; // 数据文件条数
const int D1 = 1; // 数据重复倍数，方便模拟海量数据
const int N = N1*D1;

__global__ void cal_dis(double *train_data, double *test_data, double *dis, int pitch)
{
	int tid = blockIdx.x;
	if(tid<N)
	{
		double temp = 0.0;
		double sum = 0.0;
		for(int i=0;i<D;i++)
		{
			temp = *((double*)((char*)train_data + tid * pitch) + i) - test_data[i];
			sum += temp * temp;
		}
		dis[tid] = sum;
	}
}

void print(double *data)
{
	cout<<"training data:"<<endl;
	for(int i=0;i<N;i++)
	{
		for(int j=0;j<D;j++)
		{
			cout<< fixed << setprecision(8)<<*(data+i*D+j)<<" ";
		}
		cout<<endl;
	}
}
 
void print(double *data, int n)
{
	for(int i=0;i<n;i++)
	{
		cout<< fixed << setprecision(8)<<data[i]<<" ";
	}
	cout<<endl;
}


int read_data(double *data_set)
{
	double f1;
	const char s[2] = ",";
	char *token, *line;
	FILE *fp;
	double test[D];

	// 一个数字假设占20字符，目前是保留16位小数，一共18个字符
	line = (char *)malloc(20*D*sizeof(char)); 

	fp = fopen("../vector.data" , "r");
	if(fp == NULL) {
		perror("打开文件时发生错误");
		return(-1);
	}

	// 读N+1行，最后1行做测试
	for(int i=0;i<N1+1;i++) {
		if( fgets (line, 20*D*sizeof(char), fp)!=NULL ) {
			token = strtok(line, s);

			int j = 0;
			while (token != NULL)
			{
				f1 = atof(token);
				//printf("%.8f ", f1);
				*(data_set+i*D+j)=f1;

				token = strtok(NULL, s);
				j++;
			}
			//puts("");

		} else {
			break;
		}
	}

	fclose(fp);

	free(line);

	for(int i=0;i<D;i++) test[i]=*(data_set+N1*D+i); // 保存测试向量

	for(int d=1;d<D1;d++){ // 复制数据
		for(int i=0;i<N1;i++){
			for(int j=0;j<D;j++){
				*(data_set+(N1*d+i)*D+j)= *(data_set+i*D+j);
			}
		}
	}

	for(int i=0;i<D;i++) *(data_set+N*D+i)=test[i]; // 恢复测试向量

	return 0;
}

int main()
{
	double *h_train_data, *h_test_data;
	double distance[N];
 
	double *d_train_data , *d_test_data , *d_dis;
 
	struct timeval t1,t2;
	double timeuse;

	cout<<"num= "<<N<<"\tdim= "<<D<<endl;

	h_train_data = (double*)malloc((N+1)*D*sizeof(double));
	if (h_train_data==NULL){
		puts("alloc memory fail!");
		exit(-1);
	}

	size_t pitch_d;
	size_t pitch_h = D * sizeof(double) ; 

	//allocate memory on GPU 
	hipMallocPitch( &d_train_data, &pitch_d, D*sizeof(double), N); 
	hipMalloc((void**)&d_test_data, D*sizeof(double));
	hipMalloc((void**)&d_dis, N*sizeof(double));

	//initialize training data
	read_data(h_train_data);
	//print(h_train_data);
 
	//initialize testing data
	h_test_data = h_train_data+D*N;
	cout<<"testing data:"<<endl;
	//print(h_test_data,D);
 
	gettimeofday(&t1,NULL);

	//copy training and testing data from host to device
	hipMemcpy2D(d_train_data, pitch_d, h_train_data, pitch_h, D*sizeof(double), N, hipMemcpyHostToDevice);
	hipMemcpy(d_test_data, h_test_data, D*sizeof(double), hipMemcpyHostToDevice);
 
	//calculate the distance
	cal_dis<<<N,1>>>(d_train_data,d_test_data,d_dis,pitch_d);
 
	//copy distance data from device to host
	hipMemcpy(distance, d_dis, N*sizeof(double), hipMemcpyDeviceToHost);

	gettimeofday(&t2, NULL);
 
	cout<<"distance:"<<endl;
	//print(distance, N);

	hipFree(d_train_data);
	hipFree(d_test_data);
	hipFree(d_dis);
	free(h_train_data);
	 
	timeuse = (t2.tv_sec - t1.tv_sec) + (double)(t2.tv_usec - t1.tv_usec)/1000000.0;
	cout << "[ time taken: " << fixed << setprecision(6) << timeuse << "s ]" << endl;



	return 0;
}  