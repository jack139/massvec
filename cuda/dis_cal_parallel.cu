
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <iostream>   
#include <iomanip>                                                                                                    
#include <stdio.h>
#include <stdlib.h>
#include <time.h>


using namespace std;

const int N = 10;
const int D = 10;


__global__ void cal_dis(double *train_data, double *test_data, double *dis,int pitch)
{
	int tid = blockIdx.x;
	if(tid<N)
	{
		int temp = 0;
		int sum = 0;
		for(int i=0;i<D;i++)
		{
			temp = *((int*)((char*)train_data + tid * pitch) + i) - test_data[i];
			sum += temp * temp;
		}
		dis[tid] = sum;
	}
}

void print(double *data)
{
	cout<<"training data:"<<endl;
	for(int i=0;i<N;i++)
	{
		for(int j=0;j<D;j++)
		{
			cout<< fixed << setprecision(8)<<*(data+i*D+j)<<" ";
		}
		cout<<endl;
	}
}
 
void print(double *data, int n)
{
	for(int i=0;i<n;i++)
	{
		cout<< fixed << setprecision(8)<<data[i]<<" ";
	}
	cout<<endl;
}


int read_data(double *data_set)
{
	double f1;
	const char s[2] = ",";
	char *token, *line;
	FILE *fp;

	// 一个数字假设占20字符，目前是保留16位小数，一共18个字符
	line = (char *)malloc(20*D*sizeof(char)); 

	fp = fopen("../vector.data" , "r");
	if(fp == NULL) {
		perror("打开文件时发生错误");
		return(-1);
	}

	// 读N+1行，最后1行做测试
	for(int i=0;i<N+1;i++) {
		if( fgets (line, 20*2048*sizeof(char), fp)!=NULL ) {
			token = strtok(line, s);

			int j = 0;
			while (token != NULL)
			{
				f1 = atof(token);
				//printf("%.16f ", f1);
				*(data_set+i*D+j)=f1;

				token = strtok(NULL, s);
				j++;
			}
			//puts("");

		} else {
			break;
		}
	}

	fclose(fp);

	free(line);

	return 0;
}

int main()
{
	double *h_train_data, *h_test_data;
	double distance[N];
 
	double *d_train_data , *d_test_data , *d_dis;
 
	struct timeval t1,t2;
	double timeuse;

	h_train_data = (double *)malloc((N+1)*D*sizeof(double));
	if (h_train_data==NULL){
		puts("alloc memory fail!");
		exit(-1);
	}

	size_t pitch_d;
	size_t pitch_h = D * sizeof(double) ; 

	//allocate memory on GPU 
	hipMallocPitch( &d_train_data , &pitch_d , D * sizeof(double) , N ); 
	hipMalloc( (void**)&d_test_data ,  D*sizeof(double) );
	hipMalloc( (void**)&d_dis , N*sizeof(double) );

	//initialize training data
	read_data(h_train_data);
	print(h_train_data);
 
	//initialize testing data
	h_test_data = h_train_data+D*N;
	cout<<"testing data:"<<endl;
	print(h_test_data,D);
 
	gettimeofday(&t1,NULL);

	//copy training and testing data from host to device
	hipMemcpy2D( d_train_data , pitch_d , h_train_data , pitch_h , D * sizeof(double) , N , hipMemcpyHostToDevice );
	hipMemcpy( d_test_data,  h_test_data ,  D*sizeof(double), hipMemcpyHostToDevice);
 
	//calculate the distance
	cal_dis<<<N,1>>>( d_train_data,d_test_data,d_dis,pitch_d );
 
	//copy distance data from device to host
	hipMemcpy( distance , d_dis  , N*sizeof(double) , hipMemcpyDeviceToHost);

	gettimeofday(&t2,NULL);
 
	cout<<"distance:"<<endl;
	print(distance , N);

	hipFree(d_train_data);
	hipFree(d_test_data);
	hipFree(d_dis);
	free(h_train_data);
	 
	timeuse = (t2.tv_sec - t1.tv_sec) + (double)(t2.tv_usec - t1.tv_usec)/1000000.0;
	cout << "[ time taken: " << fixed << setprecision(6) << timeuse << "s ]" << endl;



	return 0;
}  
