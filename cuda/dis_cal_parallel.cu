
#include <hip/hip_runtime.h>
#include <iostream>   
#include <iomanip>                                                                                                    
#include <stdio.h>
#include <stdlib.h>
#include <time.h>


using namespace std;

const int D = 2048;
const int N1 = 10000; // 数据文件条数
const int D1 = 90; // 数据重复倍数，方便模拟海量数据
const long N = N1*D1;


__global__ void cal_dis(float *train_data, float *test_data, float *dis, int pitch)
{
	//long tid = blockIdx.x;
	long tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid<N)
	{
		float temp = 0.0;
		float sum = 0.0;
		for(int i=0;i<D;i++)
		{
			temp = *((float*)((char*)train_data + tid * pitch) + i) - test_data[i];
			sum += temp * temp;
		}
		dis[tid] = sum;
	}
}

void print(float *data)
{
	cout<<"training data:"<<endl;
	for(long i=0;i<N;i++)
	{
		for(int j=0;j<D;j++)
		{
			cout<< fixed << setprecision(8)<<*(data+i*D+j)<<" ";
		}
		cout<<endl;
	}
}
 
void print(float *data, int n)
{
	for(int i=0;i<n;i++)
	{
		cout<< fixed << setprecision(8)<<data[i]<<" ";
	}
	cout<<endl;
}


int read_data(float *data_set)
{
	float f1;
	const char s[2] = ",";
	char *token, *line;
	FILE *fp;
	float test[D];

	// 一个数字假设占20字符，目前是保留16位小数，一共18个字符
	line = (char *)malloc(20*D*sizeof(char)); 

	fp = fopen("../vector.data" , "r");
	if(fp == NULL) {
		perror("打开文件时发生错误");
		return(-1);
	}

	// 读N+1行，最后1行做测试
	for(int i=0;i<N1+1;i++) {
		if( fgets (line, 20*D*sizeof(char), fp)!=NULL ) {
			token = strtok(line, s);

			int j = 0;
			while (token != NULL)
			{
				f1 = atof(token);
				//printf("%.8f ", f1);
				*(data_set+i*D+j)=f1;

				token = strtok(NULL, s);
				j++;
			}
			//puts("");

		} else {
			break;
		}
	}

	fclose(fp);

	free(line);

	for(int i=0;i<D;i++) test[i]=*(data_set+N1*D+i); // 保存测试向量

	for(int d=1;d<D1;d++){ // 复制数据
		for(int i=0;i<N1;i++){
			for(int j=0;j<D;j++){
				*(data_set+(N1*d+i)*D+j)= *(data_set+i*D+j);
			}
		}
	}

	for(int i=0;i<D;i++) *(data_set+N*D+i)=test[i]; // 恢复测试向量

	return 0;
}

int main()
{
	

	float *h_train_data, *h_test_data;
	float distance[N];
 
	float *d_train_data , *d_test_data , *d_dis;
 
	float time1, time2;


	// 显示GPU资源
	int dev = 0;
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, dev);
    std::cout << "使用GPU device " << dev << ": " << devProp.name << std::endl;
    std::cout << "SM的数量：" << devProp.multiProcessorCount << std::endl;
    std::cout << "每个线程块的共享内存大小：" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
    std::cout << "每个线程块的最大线程数：" << devProp.maxThreadsPerBlock << std::endl;
    std::cout << "每个EM的最大线程数：" << devProp.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "每个EM的最大线程束数：" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
    //-----------

	hipEvent_t start1, stop1, stop2;
	hipEventCreate(&start1);
	hipEventCreate(&stop1); 
	hipEventCreate(&stop2); 

	cout<<"num= "<<N<<"\tdim= "<<D<<endl;

	h_train_data = (float*)malloc((N+1)*D*sizeof(float));
	if (h_train_data==NULL){
		puts("alloc memory fail!");
		exit(-1);
	}

	size_t pitch_d;
	size_t pitch_h = D * sizeof(float) ; 

	//allocate memory on GPU 
	hipMallocPitch( &d_train_data, &pitch_d, D*sizeof(float), N); 
	hipMalloc((void**)&d_test_data, D*sizeof(float));
	hipMalloc((void**)&d_dis, (N+1)*sizeof(float)); // d_ids[N] 存最小值

	//initialize training data
	read_data(h_train_data);
	//print(h_train_data);
 
	//initialize testing data
	h_test_data = h_train_data+D*N;
	cout<<"testing data:"<<endl;
	//print(h_test_data,D);
 
	hipEventRecord(start1, 0);

	//copy training and testing data from host to device
	hipMemcpy2D(d_train_data, pitch_d, h_train_data, pitch_h, D*sizeof(float), N, hipMemcpyHostToDevice);
	hipMemcpy(d_test_data, h_test_data, D*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_dis, distance, N*sizeof(float), hipMemcpyHostToDevice);
 
	// 定义kernel的执行配置
	dim3 blockSize(1);
	dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
	printf("grid size: %d\tblock size: %d\n", gridSize.x, blockSize.x);
	// 执行kernel
	cal_dis<<<gridSize, blockSize>>>(d_train_data,d_test_data,d_dis,pitch_d);

	//calculate the distance
	//cal_dis<<<N,1>>>(d_train_data,d_test_data,d_dis,pitch_d);
 
	//copy distance data from device to host
	hipMemcpy(distance, d_dis, N*sizeof(float), hipMemcpyDeviceToHost);

	hipEventRecord(stop1, 0);

	float minimum = distance[0];
	for(long i=1;i<N;i++) if (distance[i]<minimum) minimum=distance[i];

	hipEventRecord(stop2, 0);
 
	cout<<"distance:"<<endl;
	//print(distance, N);

	hipFree(d_train_data);
	hipFree(d_test_data);
	hipFree(d_dis);
	free(h_train_data);
	
	cout << "min= " << fixed << setprecision(8) << minimum << endl;

	hipEventElapsedTime(&time1, start1, stop1);
	hipEventElapsedTime(&time2, stop1, stop2);
	printf("[ time taken: %fms %fms ]\n",time1, time2);


	return 0;
}  
