
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <iostream>   
#include <iomanip>                                                                                                    
#include <stdio.h>
#include <stdlib.h>
#include <time.h>


using namespace std;

const int N = 100;
const int D = 10;
const float MAX = 10000.0;

__global__ void cal_dis(float *train_data, float *test_data, float *dis,int pitch)
{
	int tid = blockIdx.x;
	if(tid<N)
	{
		int temp = 0;
		int sum = 0;
		for(int i=0;i<D;i++)
		{
			temp = *((int*)((char*)train_data + tid * pitch) + i) - test_data[i];
			sum += temp * temp;
		}
		dis[tid] = sum;
	}
}
 
void print(float data[][D])
{
	cout<<"training data:"<<endl;
 	for(int i=0;i<N;i++)
	{
		for(int j=0;j<D;j++)
		{
			cout<< fixed << setprecision(6)<<*(*(data+i)+j)<<" ";		
		}
		cout<<endl;
	}
}
 
void print(float *data,int n)
{
	for(int i=0;i<n;i++)
	{
		cout<< fixed << setprecision(6)<<data[i]<<" ";
	}
	cout<<endl;
}
 
int main()
{
	float h_train_data[N][D] , h_test_data[D] , distance[N];
 
	float *d_train_data , *d_test_data , *d_dis;
 
 	struct timeval t1,t2;
    double timeuse;

	size_t pitch_d;
	size_t pitch_h = D * sizeof(float) ;
 
	//allocate memory on GPU 
	hipMallocPitch( &d_train_data , &pitch_d , D * sizeof(float) , N ); 
	hipMalloc( (void**)&d_test_data ,  D*sizeof(float) );
	hipMalloc( (void**)&d_dis , N*sizeof(float) );
 
	//initialize training data
	srand( (unsigned)time(NULL) );
	for( int i=0;i<N;i++ )
	{
		for( int j=0;j<D;j++)
		{
			h_train_data[i][j] = rand()/MAX;
		}
	}
	print(h_train_data);
 
	//initialize testing data
	for( int j=0;j<D;j++ )
	{
	  	h_test_data[j] = rand()/MAX;
	}
	cout<<"testing data:"<<endl;
	print(h_test_data,D);
 
	gettimeofday(&t1,NULL);

	//copy training and testing data from host to device
	hipMemcpy2D( d_train_data , pitch_d , h_train_data , pitch_h , D * sizeof(float) , N , hipMemcpyHostToDevice );
	hipMemcpy( d_test_data,  h_test_data ,  D*sizeof(float), hipMemcpyHostToDevice);
 
	//calculate the distance
	cal_dis<<<N,1>>>( d_train_data,d_test_data,d_dis,pitch_d );
 
	//copy distance data from device to host
	hipMemcpy( distance , d_dis  , N*sizeof(float) , hipMemcpyDeviceToHost);

	gettimeofday(&t2,NULL);
 
	cout<<"distance:"<<endl;
	print(distance , N);

	hipFree(d_train_data);
	hipFree(d_test_data);
	hipFree(d_dis);
 
 	timeuse = (t2.tv_sec - t1.tv_sec) + (double)(t2.tv_usec - t1.tv_usec)/1000000.0;
  	cout << "[ time taken: " << fixed << setprecision(6) << timeuse << "s ]" << endl;

	return 0;
}  
